#include <stdio.h>

#include <algorithm>
#include <hip/hip_runtime.h>
#include <random>
#include <vector>

template <typename T>
__global__ void gemm_v4(size_t m, size_t n, size_t k, T alpha, const T *A,
                        size_t lda, const T *B, size_t ldb, T beta, T *C,
                        size_t ldc) {
  size_t C_row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  size_t C_col_idx = blockIdx.y * blockDim.y + threadIdx.y;

  if (C_row_idx < m and C_col_idx < n) {
    T sum = static_cast<T>(0);
    for (size_t k_idx = 0; k_idx < k; ++k_idx) {
      sum += A[C_row_idx * lda + k_idx] * B[k_idx * ldb + C_col_idx];
    }
    C[C_row_idx * ldc + C_col_idx] = alpha * sum + beta * C[C_row_idx * ldc + C_col_idx];
  }
}

namespace gemm {
template <typename T>
void launch_gemm_v4(size_t m, size_t n, size_t k, const T* alpha, const T *A,
                    size_t lda, const T *B, size_t ldb, const T* beta, T *C,
                    size_t ldc, hipStream_t stream) {
  const dim3 block_dim{32U, 32U, 1U};
  const dim3 grid_dim{
      (static_cast<unsigned int>(m) + block_dim.x - 1U) / block_dim.x,
      (static_cast<unsigned int>(n) + block_dim.y - 1U) / block_dim.y, 1U};
  gemm_v4<T><<<grid_dim, block_dim, 0U, stream>>>(m, n, k, *alpha, A, lda, B, ldb,
                                                 *beta, C, ldc);
}
template void launch_gemm_v4<float>(size_t m, size_t n, size_t k, const float* alpha,
                                    const float *A, size_t lda, 
                                    const float *B, size_t ldb, const float* beta, 
                                    float *C, size_t ldc, hipStream_t stream);

template void launch_gemm_v4<double>(size_t m, size_t n, size_t k, const double* alpha,
                                     const double *A, size_t lda,
                                     const double *B, size_t ldb, const double* beta,
                                     double *C, size_t ldc, hipStream_t stream);
} // namespace gemm